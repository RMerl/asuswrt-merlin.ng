#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cuda/vector_helpers.cuh"

template<typename T>
__device__ inline void Subsample_Nearest(hipTextureObject_t tex,
                                         T *dst,
                                         int dst_width, int dst_height, int dst_pitch,
                                         int src_width, int src_height,
                                         int bit_depth)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        dst[yo*dst_pitch+xo] = tex2D<T>(tex, xi, yi);
    }
}

template<typename T>
__device__ inline void Subsample_Bilinear(hipTextureObject_t tex,
                                          T *dst,
                                          int dst_width, int dst_height, int dst_pitch,
                                          int src_width, int src_height,
                                          int bit_depth)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        intT r = { 0 };
        vec_set_scalar(r, 2);
        r += tex2D<T>(tex, xi - dx, yi - dy);
        r += tex2D<T>(tex, xi + dx, yi - dy);
        r += tex2D<T>(tex, xi - dx, yi + dy);
        r += tex2D<T>(tex, xi + dx, yi + dy);
        vec_set(dst[yo*dst_pitch+xo], r >> 2);
    }
}

extern "C" {

#define NEAREST_KERNEL(T) \
    __global__ void Subsample_Nearest_ ## T(hipTextureObject_t src_tex,                  \
                                            T *dst,                                       \
                                            int dst_width, int dst_height, int dst_pitch, \
                                            int src_width, int src_height,                \
                                            int bit_depth)                                \
    {                                                                                     \
        Subsample_Nearest<T>(src_tex, dst,                                                \
                              dst_width, dst_height, dst_pitch,                           \
                              src_width, src_height,                                      \
                              bit_depth);                                                 \
    }

NEAREST_KERNEL(uchar)
NEAREST_KERNEL(uchar2)
NEAREST_KERNEL(uchar4)

NEAREST_KERNEL(ushort)
NEAREST_KERNEL(ushort2)
NEAREST_KERNEL(ushort4)

#define BILINEAR_KERNEL(T) \
    __global__ void Subsample_Bilinear_ ## T(hipTextureObject_t src_tex,                  \
                                             T *dst,                                       \
                                             int dst_width, int dst_height, int dst_pitch, \
                                             int src_width, int src_height,                \
                                             int bit_depth)                                \
    {                                                                                      \
        Subsample_Bilinear<T>(src_tex, dst,                                                \
                              dst_width, dst_height, dst_pitch,                            \
                              src_width, src_height,                                       \
                              bit_depth);                                                  \
    }

BILINEAR_KERNEL(uchar)
BILINEAR_KERNEL(uchar2)
BILINEAR_KERNEL(uchar4)

BILINEAR_KERNEL(ushort)
BILINEAR_KERNEL(ushort2)
BILINEAR_KERNEL(ushort4)

}
