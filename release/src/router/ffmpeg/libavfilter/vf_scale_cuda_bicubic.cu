#include "hip/hip_runtime.h"
/*
 * This file is part of FFmpeg.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cuda/vector_helpers.cuh"
#include "vf_scale_cuda.h"

typedef float4 (*coeffs_function_t)(float, float);

__device__ inline float4 lanczos_coeffs(float x, float param)
{
    const float pi = 3.141592654f;

    float4 res = make_float4(
        pi * (x + 1),
        pi * x,
        pi * (x - 1),
        pi * (x - 2));

    res.x = res.x == 0.0f ? 1.0f :
        __sinf(res.x) * __sinf(res.x / 2.0f) / (res.x * res.x / 2.0f);
    res.y = res.y == 0.0f ? 1.0f :
        __sinf(res.y) * __sinf(res.y / 2.0f) / (res.y * res.y / 2.0f);
    res.z = res.z == 0.0f ? 1.0f :
        __sinf(res.z) * __sinf(res.z / 2.0f) / (res.z * res.z / 2.0f);
    res.w = res.w == 0.0f ? 1.0f :
        __sinf(res.w) * __sinf(res.w / 2.0f) / (res.w * res.w / 2.0f);

    return res / (res.x + res.y + res.z + res.w);
}

__device__ inline float4 bicubic_coeffs(float x, float param)
{
    const float A = param == SCALE_CUDA_PARAM_DEFAULT ? 0.0f : -param;

    float4 res;
    res.x = ((A * (x + 1) - 5 * A) * (x + 1) + 8 * A) * (x + 1) - 4 * A;
    res.y = ((A + 2) * x - (A + 3)) * x * x + 1;
    res.z = ((A + 2) * (1 - x) - (A + 3)) * (1 - x) * (1 - x) + 1;
    res.w = 1.0f - res.x - res.y - res.z;

    return res;
}

__device__ inline void derived_fast_coeffs(float4 coeffs, float x, float *h0, float *h1, float *s)
{
    float g0 = coeffs.x + coeffs.y;
    float g1 = coeffs.z + coeffs.w;

    *h0 = coeffs.y / g0 - 0.5f;
    *h1 = coeffs.w / g1 + 1.5f;
    *s  = g0 / (g0 + g1);
}

template<typename V>
__device__ inline V apply_coeffs(float4 coeffs, V c0, V c1, V c2, V c3)
{
    V res = c0 * coeffs.x;
    res  += c1 * coeffs.y;
    res  += c2 * coeffs.z;
    res  += c3 * coeffs.w;

    return res;
}

template<typename T>
__device__ inline void Subsample_Bicubic(coeffs_function_t coeffs_function,
                                         hipTextureObject_t src_tex,
                                         T *dst,
                                         int dst_width, int dst_height, int dst_pitch,
                                         int src_width, int src_height,
                                         int bit_depth, float param)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale - 0.5f;
        float yi = (yo + 0.5f) * vscale - 0.5f;
        float px = floor(xi);
        float py = floor(yi);
        float fx = xi - px;
        float fy = yi - py;

        float factor = bit_depth > 8 ? 0xFFFF : 0xFF;

        float4 coeffsX = coeffs_function(fx, param);
        float4 coeffsY = coeffs_function(fy, param);

#define PIX(x, y) tex2D<floatT>(src_tex, (x), (y))

        dst[yo * dst_pitch + xo] = from_floatN<T, floatT>(
            apply_coeffs<floatT>(coeffsY,
                apply_coeffs<floatT>(coeffsX, PIX(px - 1, py - 1), PIX(px, py - 1), PIX(px + 1, py - 1), PIX(px + 2, py - 1)),
                apply_coeffs<floatT>(coeffsX, PIX(px - 1, py    ), PIX(px, py    ), PIX(px + 1, py    ), PIX(px + 2, py    )),
                apply_coeffs<floatT>(coeffsX, PIX(px - 1, py + 1), PIX(px, py + 1), PIX(px + 1, py + 1), PIX(px + 2, py + 1)),
                apply_coeffs<floatT>(coeffsX, PIX(px - 1, py + 2), PIX(px, py + 2), PIX(px + 1, py + 2), PIX(px + 2, py + 2))
            ) * factor
        );

#undef PIX
    }
}

/* This does not yield correct results. Most likely because of low internal precision in tex2D linear interpolation */
template<typename T>
__device__ inline void Subsample_FastBicubic(coeffs_function_t coeffs_function,
                                             hipTextureObject_t src_tex,
                                             T *dst,
                                             int dst_width, int dst_height, int dst_pitch,
                                             int src_width, int src_height,
                                             int bit_depth, float param)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale - 0.5f;
        float yi = (yo + 0.5f) * vscale - 0.5f;
        float px = floor(xi);
        float py = floor(yi);
        float fx = xi - px;
        float fy = yi - py;

        float factor = bit_depth > 8 ? 0xFFFF : 0xFF;

        float4 coeffsX = coeffs_function(fx, param);
        float4 coeffsY = coeffs_function(fy, param);

        float h0x, h1x, sx;
        float h0y, h1y, sy;
        derived_fast_coeffs(coeffsX, fx, &h0x, &h1x, &sx);
        derived_fast_coeffs(coeffsY, fy, &h0y, &h1y, &sy);

#define PIX(x, y) tex2D<floatT>(src_tex, (x), (y))

        floatT pix[4] = {
            PIX(px + h0x, py + h0y),
            PIX(px + h1x, py + h0y),
            PIX(px + h0x, py + h1y),
            PIX(px + h1x, py + h1y)
        };

#undef PIX

        dst[yo * dst_pitch + xo] = from_floatN<T, floatT>(
            lerp_scalar(
                lerp_scalar(pix[3], pix[2], sx),
                lerp_scalar(pix[1], pix[0], sx),
                sy) * factor
        );
    }
}

extern "C" {

#define BICUBIC_KERNEL(T) \
    __global__ void Subsample_Bicubic_ ## T(hipTextureObject_t src_tex,                  \
                                            T *dst,                                       \
                                            int dst_width, int dst_height, int dst_pitch, \
                                            int src_width, int src_height,                \
                                            int bit_depth, float param)                   \
    {                                                                                     \
        Subsample_Bicubic<T>(&bicubic_coeffs, src_tex, dst,                               \
                             dst_width, dst_height, dst_pitch,                            \
                             src_width, src_height,                                       \
                             bit_depth, param);                                           \
    }

BICUBIC_KERNEL(uchar)
BICUBIC_KERNEL(uchar2)
BICUBIC_KERNEL(uchar4)

BICUBIC_KERNEL(ushort)
BICUBIC_KERNEL(ushort2)
BICUBIC_KERNEL(ushort4)


#define LANCZOS_KERNEL(T) \
    __global__ void Subsample_Lanczos_ ## T(hipTextureObject_t src_tex,                  \
                                            T *dst,                                       \
                                            int dst_width, int dst_height, int dst_pitch, \
                                            int src_width, int src_height,                \
                                            int bit_depth, float param)                   \
    {                                                                                     \
        Subsample_Bicubic<T>(&lanczos_coeffs, src_tex, dst,                               \
                             dst_width, dst_height, dst_pitch,                            \
                             src_width, src_height,                                       \
                             bit_depth, param);                                           \
    }

LANCZOS_KERNEL(uchar)
LANCZOS_KERNEL(uchar2)
LANCZOS_KERNEL(uchar4)

LANCZOS_KERNEL(ushort)
LANCZOS_KERNEL(ushort2)
LANCZOS_KERNEL(ushort4)

}
